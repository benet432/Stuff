#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{
__global__ void 
rtruncnorm_kernel(float *vals, int n, 
                  float *mu, float *sigma, 
                  float *lo, float *hi,
                  int rng_a, int rng_b,
                  int rng_c,
                  int maxtries)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    
    if (idx < n)
    {
            // Setup the RNG:
            hiprandState rng;
            hiprand_init (rng_a+idx*rng_b, rng_c, 0, &rng);
            // Sample:
            int accept=0;
            int numtries=0;
            while (!accept && numtries < maxtries)
            {
                  numtries ++;
                  vals[idx]=mu[idx]+sigma[idx]*hiprand_normal(&rng);
                  if (vals[idx]>=lo[idx] && vals[idx]<=hi[idx])
                  { accept=1; }
                  else {}
            }
                     
    }
    return;
}

} // END extern "C"